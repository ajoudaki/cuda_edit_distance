
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <cassert>
#include <ctime>

#define DEBUG

typedef float dist_t;
typedef dist_t result_t;

// Convenience function for checking CUDA runtime API results
inline
hipError_t checkCuda(hipError_t result)
{
//#if defined(DEBUG) || defined(_DEBUG)
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n",
                hipGetErrorString(result));
        assert(result == hipSuccess);
    }
//#endif
    return result;
}

#define MIN(x,y) ((x) < (y) ? (x) : (y)) //calculate minimum between two values


__global__ void upper_diagonal(const char *seq1, const char *seq2, unsigned int d, result_t *d0, result_t* d1, result_t* d2) {
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<=0 or i >= d) {
        d2[i] = d;
    } else {
        result_t t = seq1[i-1]!=seq2[d - i - 1];
        d2[i] = MIN(t + d0[i-1], MIN(d1[i],d1[i-1])+1 );
    }
}

// TODO debug the lower dimensional
__global__ void lower_diagonal(const char *seq1, const char *seq2, unsigned int len, unsigned int d, result_t *d0, result_t* d1, result_t* d2) {
    int off = d - len, off2 = (d != len + 1);// diagonal: d
    auto i = blockIdx.x * blockDim.x + threadIdx.x + off;

    result_t t = seq1[i-1]!=seq2[d-i-1];
    d2[i] = MIN(t + d0[i + off2], MIN(d1[i],d1[i+1])+1 );
}

void levenshtein_distance(char *s1, char *s2, unsigned int l1, unsigned int l2, std::vector<std::vector<unsigned int>> &dist) {
    dist = std::vector<std::vector<unsigned int>>(l2+1, std::vector<unsigned int>(l1+1, (l1+l2)*2));
    for(int i=0;i<=l1;i++) {
        dist[0][i] = i;
    }
    for(int j=0;j<=l2;j++) {
        dist[j][0] = j;
    }
    for (int j=1;j<=l1;j++) {
        for(int i=1;i<=l2;i++) {
            int track, t;
            if(s1[i-1] == s2[j-1]) {
                track= 0;
            } else {
                track = 1;
            }
            t = MIN((dist[i-1][j]+1),(dist[i][j-1]+1));
            dist[i][j] = MIN(t,(dist[i-1][j-1]+track));
        }
    }
#if defined(DEBUG) || defined(_DEBUG)
    std::cout<<"The Levinstein distance is:"<<dist[l2][l1] << std::endl;
#endif
}

int main() {
    uint64_t blocksize = 1;
    uint64_t len = (1<<2);
    unsigned int alphpabet_size = 20;

    // load sequences into pinnable memory
    char *h1, *h2;
    auto bytes = (len+1) * sizeof(char); // zero-pad by 1 byte to add end-of-sequence-char:0
    checkCuda(hipHostMalloc(&h1, bytes, hipHostMallocDefault));
    checkCuda(hipHostMalloc(&h2, bytes , hipHostMallocDefault));
    for (int i=0; i < len; i++) {
        int r = rand() ;
        h1[i] = (char)(r % alphpabet_size + (int)'A');
        r = rand();
        h2[i]= (char)(r % alphpabet_size + (int)'A') ;
    }
    h1[len] = h2[len] = 0;

    // move to device memory
    char *s1, *s2;
    checkCuda(hipMalloc(&s1, bytes) ) ;
    checkCuda( hipMalloc(&s2, bytes ) );
    checkCuda( hipMemcpy(s1, h1, bytes, hipMemcpyHostToDevice) );
    checkCuda( hipMemcpy(s2, h2, bytes, hipMemcpyHostToDevice) );

    std::clock_t c_start, c_end;


    result_t *mem, *diag0, *diag1, *diag2, *tmp;
    uint64_t pitch = len + blocksize;                       // pad distances by at least blocksize
    bytes = 3* pitch * sizeof(result_t) ;                   // allocate 3 x pitch * size type
    checkCuda( hipMalloc(&mem, bytes) );       // allocate on unified memory
    checkCuda(hipMemset(mem, 2* len, bytes));   // initialize distances to maximum value 2*len
    diag0 = mem;
    diag1 = diag0 + pitch;
    diag2 = diag1 + pitch;

    c_start = std::clock();
    printf("Computing the edit distance blocks... \n" );


    uint64_t d;
    for (d=0; d <= len; d++) {
        upper_diagonal<<< (d + blocksize) / blocksize, blocksize >>>(s1, s2, d, diag0, diag1, diag2);
        checkCuda( hipDeviceSynchronize() );
        tmp = diag0; diag0 = diag1; diag1 = diag2; diag2 = tmp;
    }

    for (d=len+1; d <= 2*len; d++) {
        lower_diagonal<<< (2*len-d  + blocksize) / blocksize, blocksize >>>(s1, s2, len, d, diag0, diag1, diag2);
        checkCuda( hipDeviceSynchronize() );
        tmp = diag0; diag0 = diag1; diag1 = diag2; diag2 = tmp;
    }
    dist_t *d_host;
    bytes = (len+1)* sizeof (dist_t);
    checkCuda( hipHostMalloc(&d_host, bytes, hipHostMallocDefault) );
    checkCuda( hipMemcpy(d_host, diag1, bytes, hipMemcpyDeviceToHost) );
    checkCuda(hipMemset(mem, 2* len, bytes));

    printf("edit dist CUDA: %f\n", d_host[0]);
    auto gpu_time = 1.0 * (std::clock()-c_start) / CLOCKS_PER_SEC;
    printf("GPU time: %f s\n", gpu_time);

#if defined(DEBUG) || defined(_DEBUG)
    // check results
        std::vector<std::vector<unsigned int>> dist;
        c_start = std::clock();

        levenshtein_distance(h1, h2, len, len, dist);

        auto cpu_time = 1.0 * (std::clock()-c_start) / CLOCKS_PER_SEC;
        printf("CPU time: %f s\n", cpu_time);

        printf("speed up %f\n", cpu_time/gpu_time);
        int mismatch = 0;
        for (int i=0; i<=len; i++) {
            if (d_host[i] != dist[i][len-i]) {
                mismatch++;
            }
        }
        printf("number of mismatches = %d, out of %d\n",mismatch, len);
        assert(mismatch==0 && " mismatch found ");
#endif

#if defined(_DEBUG)
    printf("final dist:\n");
        for (int i=0; i<=len; i++) {
            printf("%d ",d_host[i]);
        }
        printf("\nED diag %lu =\n",len);
        for (int i=0; i<=len; i++) {
            printf("%d,",dist[i][len-i]);
        }
        printf("\n");
        printf("%.32s...,%.32s...\n", h1, h2);

#endif

}
