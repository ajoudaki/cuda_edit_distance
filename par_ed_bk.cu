#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <cassert>
#include <ctime>

#define DEBUG 0
typedef float dist_t;
typedef dist_t result_t;

// Convenience function for checking CUDA runtime API results
inline
hipError_t checkCuda(hipError_t result)
{
//#if defined(DEBUG) || defined(_DEBUG)
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n",
                hipGetErrorString(result));
        assert(result == hipSuccess);
    }
//#endif
    return result;
}

#define MIN(x,y) ((x) < (y) ? (x) : (y)) //calculate minimum between two values



__global__ void compute_diagonal(char *seq1, char *seq2, unsigned int l, result_t *d0, result_t* d1, result_t* d2) {
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if (DEBUG>=2)
        printf(">> l=%d, i=%d, blockdim=%d, blockindex=%d\n", l, i, blockDim.x, blockIdx.x);

    if (i<=0 or i>=l) {
        d2[i] = l;
        if (DEBUG>=2)
            printf(">> i=%d, j=%d, d_%d[%d]=%d,\n", i,l-i, l, i, d2[i]);
    } else {
        result_t t = seq1[i-1]!=seq2[l-i-1];
        if (DEBUG>=2)
            printf(">> i=%d, j=%d, t=%d, d_%d[%d]=%d, d_%d[%d]=%d, d_%d[%d]=%d,\n", i, l-i, t, l, i, d2[i], l-1, i, d1[i], l-1, i-1, d1[i-1]);
        d2[i] = MIN(t + d0[i-1], MIN(d1[i],d1[i-1])+1 );
    }
}

void levenshtein_distance(char *s1, char *s2, unsigned int l1, unsigned int l2, std::vector<std::vector<unsigned int>> &dist) {
    // init dist = (l2+1)x(l1+1)
    dist = std::vector<std::vector<unsigned int>>(l2+1, std::vector<unsigned int>(l1+1, (l1+l2)*2));
    for(int i=0;i<=l1;i++) {
        dist[0][i] = i;
    }
    for(int j=0;j<=l2;j++) {
        dist[j][0] = j;
    }
    for (int j=1;j<=l1;j++) {
        for(int i=1;i<=l2;i++) {
            int track, t;
            if(s1[i-1] == s2[j-1]) {
                track= 0;
            } else {
                track = 1;
            }
            t = MIN((dist[i-1][j]+1),(dist[i][j-1]+1));
            dist[i][j] = MIN(t,(dist[i-1][j-1]+track));
        }
    }
    if (DEBUG>=1)
        std::cout<<"The Levinstein distance is:"<<dist[l2][l1] << std::endl;
}

int main() {
    uint64_t blocksize = 128;
    uint64_t len = (1<<18);
    unsigned int alphpabet_size = 4;

    // load sequences into pinnable memory
    char *h1, *h2;
    auto bytes = (len+1) * sizeof(char); // zero-pad by 1 byte to add end-of-sequence-char:0
    checkCuda(hipHostMalloc(&h1, bytes));
    checkCuda(hipHostMalloc(&h2, bytes ));
    for (int i=0; i < len; i++) {
        int r = rand() ;
        h1[i] = (char)(r % alphpabet_size + (int)'A');
        r = rand();
        h2[i]= (char)(r % alphpabet_size + (int)'A') ;
    }
    h1[len] = h2[len] = 0;

    // move to device memory
    char *s1, *s2;
    checkCuda(hipMalloc(&s1, bytes) ) ;
    checkCuda( hipMalloc(&s2, bytes ) );
    checkCuda( hipMemcpy(s1, h1, bytes, hipMemcpyHostToDevice) );
    checkCuda( hipMemcpy(s2, h2, bytes, hipMemcpyHostToDevice) );

    std::clock_t c_start, c_end;


    result_t *d0, *d1, *d2, *tmp;
    uint64_t pitch = len + blocksize;                       // pad distances by at least blocksize
    bytes = 3* pitch * sizeof(result_t) ;                   // allocate 3 x pitch * size type
    checkCuda( hipMalloc(&d0, bytes) );       // allocate on unified memory
    checkCuda(hipMemset(d0, 2* len, bytes));   // initialize distances to maximum value 2*len
    d1 = d0 + pitch;
    d2 = d1 + pitch;

    c_start = std::clock();
    printf("Computing the edit distance blocks... \n" );


    uint64_t l;
    for (l=0; l<=len; l++) {
//        d2[0] = l;
//        d2[l] = l;

        if (l<=blocksize) {
            compute_diagonal<<< l + 1, 1 >>>(s1, s2, l, d0, d1, d2);
        }  else {
            compute_diagonal<<< (l + blocksize) / blocksize, blocksize >>>(s1, s2, l, d0, d1, d2);
        }
        checkCuda( hipDeviceSynchronize() );
        tmp = d0; d0 = d1; d1 = d2; d2 = tmp;
        if (DEBUG>=2) {
            printf("d_%d = ", l);
            for (int i=0; i<=l; i++) {
                printf("%d,",d1[i]);
            }
            printf("\n");
        }
    }
    dist_t *d_host;
    bytes = (len+1)* sizeof (dist_t);
    hipHostMalloc(&d_host, bytes);
    checkCuda( hipMemcpy(d_host, d1, bytes, hipMemcpyHostToDevice) );

    auto gpu_time = 1.0 * (std::clock()-c_start) / CLOCKS_PER_SEC;
    printf("GPU time: %f s\n", gpu_time);

    if (DEBUG >= 1) {
        // check results
        std::vector<std::vector<unsigned int>> dist;
        c_start = std::clock();

        levenstein_distance(h1, h2, len, len, dist);

        auto cpu_time = 1.0 * (std::clock()-c_start) / CLOCKS_PER_SEC;
        printf("CPU time: %f s\n", cpu_time);

        printf("speed up %f\n", cpu_time/gpu_time);
        int mismatch = 0;
        for (int i=0; i<=len; i++) {
            if (d_host[i] != dist[i][len-i]) {
                mismatch++;
            }
        }
        printf("number of mismatches = %d, out of %d\n",mismatch, len);
//        assert(mismatch==0 && " mismatch found ");

        if (DEBUG>=2) {
            printf("final dist:\n");
            for (int i=0; i<=len; i++) {
                printf("%d ",d_host[i]);
            }
            printf("\nED diag %lu =\n",len);
            for (int i=0; i<=len; i++) {
                printf("%d,",dist[i][len-i]);
            }
            printf("\n");
            printf("%.32s...,%.32s...\n", h1, h2);

        }
    }


}
